#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void matmul(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    C[i * n + j] = 0;
    for (int k = 0; k < n; k++)
    {
        C[i * n + j] += A[i * n + k] * B[k * n + j];
    }
}

int main()
{
    srand(time(NULL));

    int n = 1000;
    size_t size = sizeof(float) * n * n;

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    for (int i = 0; i < n * n; i++)
    {
        A[i] = (float)rand() / (float)(RAND_MAX);
        B[i] = (float)rand() / (float)(RAND_MAX);
    }

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    clock_t t;
    t = clock();

    matmul<<<n, n>>>(d_A, d_B, d_C, n);

    t = clock() - t;
    double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
    printf("\nC=A@B took %f seconds to execute \n", time_taken);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
